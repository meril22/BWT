#include "hip/hip_runtime.h"
#include "bwt.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

int comp_size = 1;                      

int lex_compare(const void * a, const void * b)
{
    unsigned char *x1 = *(unsigned char**)a;
    unsigned char *x2 = *(unsigned char**)b;

    return memcmp(x1, x2, comp_size*sizeof(unsigned char));
}


int lex_compare_2(const void * a, const void * b)
{
    unsigned char *x1 = *(unsigned char**)a;
    unsigned char *x2 = *(unsigned char**)b;
    
    
    for (int tmp_size = comp_size; tmp_size > 0; tmp_size--){
        if(!(*x1 ^ *x2)){
            x1++;
            x2++;
        }
        else if(*x1 < *x2){
            return -1;
        }
        else{
            return 1;
        }
    }
    return 0;
}


void __global__ bwt_encode(unsigned char ** bwt_in, unsigned char ** bwt_out, int len)
{
    unsigned char ** ptr_rotations, *concat_input;                         
    ptr_rotations = (unsigned char**) malloc(len*sizeof(unsigned char*));
    concat_input = (unsigned char*)malloc(2*len*sizeof(unsigned char) + 1);
    memcpy(concat_input, *bwt_in, len*sizeof(unsigned char));
    memcpy(concat_input + len*sizeof(unsigned char), *bwt_in, len*sizeof(unsigned char));
    
    concat_input[2*len] = '\0';                                
                                                                
    int i;
    for(i = 0; i < len; i++){
        ptr_rotations[i] = &(concat_input[i]);
    }
    
    comp_size = len;                                            
    qsort(ptr_rotations, len, sizeof(unsigned char*), lex_compare);      
   
    for( i = 0; i < len ; i++){
        (*bwt_out)[i] = *(ptr_rotations[i] + (len-1)*sizeof(unsigned char));    
        if(ptr_rotations[i] == concat_input){                                   
            (*bwt_out)[len] = i/(256*256);                                      
            (*bwt_out)[len + 1] = (i%(256*256))/256;                            
            (*bwt_out)[len + 2] = (i%(256*256))%256;                            
        }
    }
    cudafree(concat_input);
    concat_input = NULL;

    cudafree(ptr_rotations);
    ptr_rotations = NULL;

    return;
}


